#include "hip/hip_runtime.h"
#include "mat.h"
#include <chrono>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#define M 3      // match
#define MM -3    // mismatch
#define W -2     // gap score
#define max(a, b) (((a) > (b)) ? (a) : (b)) // return maximum of two values
#define min(a, b) (((a) < (b)) ? (a) : (b)) // return minimum of two values



void read_sequence_from_file(const std::string& filename, std::vector<char>& seq, int line_number) {
    std::ifstream file(filename);
    if (file.is_open()) {
        std::string line;
        for (int i = 0; i < line_number; ++i) {
            if (!std::getline(file, line)) {
                std::cerr << "Error reading line " << line_number << " from file: " << filename << std::endl;
                file.close();
                return;
            }
        }
        seq.assign(line.begin(), line.end());
        file.close();
    } else {
        std::cerr << "Unable to open file: " << filename << std::endl;
    }
}

std::pair<int, int> fill_cpu(Matrix h, Matrix d, char seqA[], char seqB[]) {

  int full_max_id = 0;
  int full_max_val = 0;

  for (int i = 1; i < h.height; i++) {
    for (int j = 1; j < h.width; j++) {

      // scores
      int max_score = 0;
      int direction = 0;
      int tmp_score;
      int sim_score;

      // comparison positions
      int id = i * h.width + j;                  // current cell
      int abov_id = (i - 1) * h.width + j;       // above cell, 1
      int left_id = i * h.width + (j - 1);       // left cell, 2
      int diag_id = (i - 1) * h.width + (j - 1); // upper-left diagonal cell, 3

      // above cell
      tmp_score = h.elements[abov_id] + W;
      if (tmp_score > max_score) {
        max_score = tmp_score;
        direction = 1;
      }

      // left cell
      tmp_score = h.elements[left_id] + W;
      if (tmp_score > max_score) {
        max_score = tmp_score;
        direction = 2;
      }

      // diagonal cell (preferred)
      char baseA = seqA[j - 1];
      char baseB = seqB[i - 1];
      if (baseA == baseB) {
        sim_score = M;
      } else {
        sim_score = MM;
      }

      tmp_score = h.elements[diag_id] + sim_score;
      if (tmp_score >= max_score) {
        max_score = tmp_score;
        direction = 3;
      }

      // assign scores and direction
      h.elements[id] = max_score;
      d.elements[id] = direction;

      if (max_score > full_max_val) {
        full_max_id = id;
        full_max_val = max_score;
      }
    }
  }
  return std::make_pair(full_max_id, full_max_val);
}


void traceback(Matrix d, int max_id, char seqA[], char seqB[],
               std::vector<char> &seqA_aligned,
               std::vector<char> &seqB_aligned) {

  int max_i = max_id / d.width;
  int max_j = max_id % d.width;

  // traceback algorithm from maximum score to 0
  while (max_i > 0 && max_j > 0) {

    int id = max_i * d.width + max_j;
    int dir = d.elements[id];

    switch (dir) {
    case 1:
      --max_i;
      seqA_aligned.push_back('-');
      seqB_aligned.push_back(seqB[max_i]);
      break;
    case 2:
      --max_j;
      seqA_aligned.push_back(seqA[max_j]);
      seqB_aligned.push_back('-');
      break;
    case 3:
      --max_i;
      --max_j;
      seqA_aligned.push_back(seqA[max_j]);
      seqB_aligned.push_back(seqB[max_i]);
      break;
    case 0:
      max_i = -1;
      max_j = -1;
      break;
    }
  }
}

// print aligned sequnces
void io_seq(std::vector<char> &seqA_aligned, std::vector<char> &seqB_aligned) {

  std::cout << "Aligned sub-sequences of A and B: " << std::endl;
  int align_len = seqA_aligned.size();
  std::cout << "   ";
  for (int i = 0; i < align_len + 1; ++i) {
    std::cout << seqA_aligned[align_len - i];
  }
  std::cout << std::endl;

  std::cout << "   ";
  for (int i = 0; i < align_len + 1; ++i) {
    std::cout << seqB_aligned[align_len - i];
  }
  std::cout << std::endl << std::endl;
}



void smith_water_cpu(Matrix h, Matrix d, char seqA[], char seqB[]) {

  // populate scoring and direction matrix and find id of max score
  std::pair<int, int> result = fill_cpu(h, d, seqA, seqB);
  int max_id = result.first;
  int score = result.second;
  // traceback
  std::vector<char> seqA_aligned;
  std::vector<char> seqB_aligned;
  traceback(d, max_id, seqA, seqB, seqA_aligned, seqB_aligned);

  // print aligned sequences
  io_seq(seqA_aligned, seqB_aligned);

  std::cout << std::endl;
  std::cout << "CPU result: " << std::endl;
  std::cout << "Max score:" << score << std::endl;
}

char* vectorToCharArray(const std::vector<char>& vec) {
    char* arr = new char[vec.size() + 1]; // +1 để thêm ký tự null
    std::copy(vec.begin(), vec.end(), arr);
    arr[vec.size()] = '\0'; // thêm ký tự null vào cuối mảng
    return arr;
}

int main() {
  std::vector<char> seqA;
  read_sequence_from_file("D:\\Gpu-SW\\src\\a.txt", seqA, 1);
  std::vector<char> seqB;
  read_sequence_from_file("D:\\Gpu-SW\\src\\b.txt", seqB, 1);
  std::cout << "Seq A with length " << seqA.size() << " is: ";
  for (int i = 0; i < seqA.size(); i++)
    std::cout << seqA[i];
  std::cout << std::endl;
  std::cout << "Seq B with length " << seqB.size() << " is: ";
  for (int i = 0; i < seqB.size(); i++)
    std::cout << seqB[i];
  std::cout << std::endl;
  char* arrA = vectorToCharArray(seqA);
  char* arrB = vectorToCharArray(seqB);
  // initialize scoring and direction matrices
  Matrix scr_cpu(seqA.size() + 1, seqB.size() + 1); // cpu score matrix
  Matrix dir_cpu(seqA.size() + 1, seqB.size() + 1); // cpu direction
  Matrix scr_gpu(seqA.size() + 1, seqB.size() + 1); // gpu score matrix
  Matrix dir_gpu(seqA.size() + 1, seqB.size() + 1); // gpu direction matrix

  // apply initial condition of 0
  for (int i = 0; i < scr_cpu.height; i++) {
    for (int j = 0; j < scr_cpu.width; j++) {
      int id = i * scr_cpu.width + j;
      scr_cpu.elements[id] = 0;
      dir_cpu.elements[id] = 0;
      scr_gpu.elements[id] = 0;
      dir_gpu.elements[id] = 0;
    }
  }

  // visualize initial scoring matrix
  // io_score(std::string("init.dat"), scr_cpu, seqA, seqB);

  // CPU
  auto start_cpu = std::chrono::steady_clock::now();
  smith_water_cpu(scr_cpu, dir_cpu, arrA, arrB); // call CPU smith water
  auto end_cpu = std::chrono::steady_clock::now();
  auto diff = end_cpu - start_cpu;
  std::cout << "   CPU time = "
            << std::chrono::duration<double, std::milli>(diff).count() << " ms"
            << std::endl;
  std::cout << std::endl;

  // GPU
  // smith_water_gpu(scr_gpu, dir_gpu, seqA, seqB); // call GPU smith water
  // deallocate memory
  scr_cpu.cpu_deallocate();
  dir_cpu.cpu_deallocate();
  scr_gpu.cpu_deallocate();
  dir_gpu.cpu_deallocate();

  return 0;
}

